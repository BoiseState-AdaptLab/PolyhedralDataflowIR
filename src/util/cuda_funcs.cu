//
// Created by edavis on 12/16/16.
//

#include "cuda_funcs.h"

inline void _gpuAssert(hipError_t code, const char *file, int line) {
    if (code != hipSuccess)  {
        fprintf(stderr, "gpuAssert: %s %s %d\n", hipGetErrorString(code), file, line);
        exit(code);
    }
}

void _gpuCheck(const char* msg, const char* file, int line) {
    hipError_t _err = hipGetLastError();
    if (_err != hipSuccess) {
        fprintf(stderr, "FATAL: %s (%s at %s:%d)\n", msg, hipGetErrorString(_err), file, line);
    }
}

cuda_t *cuda_new() {
    cuda_t *cuda = (cuda_t*) calloc(1, sizeof(cuda_t));
    cuda->ndevices = 0;
    cuda->devid = 0;
    cuda->dataptr = NULL;
    cuda->datasize = 0;
    cuda->start = NULL;
    cuda->stop = NULL;
    cuda->managed = 0;
    return cuda;
}

void cuda_del(cuda_t *cuda) {
    if (cuda->start != NULL) {
        gpuAssert(hipEventDestroy(cuda->start));
    }
    if (cuda->stop != NULL) {
        gpuAssert(hipEventDestroy(cuda->stop));
    }
    if (cuda->dataptr != NULL) {
        gpuAssert(hipFree(cuda->dataptr));
    }
    free(cuda);
}

int cuda_device_count(cuda_t *cuda) {
    if (cuda->ndevices < 1) {
        gpuAssert(hipGetDeviceCount(&cuda->ndevices));
    }
    return cuda->ndevices;
}

int cuda_device_id(cuda_t *cuda) {
    return cuda->devid;
}

hipDeviceProp_t cuda_device_props(cuda_t *cuda) {
    gpuAssert(hipGetDeviceProperties(&cuda->props, cuda->devid));
    return cuda->props;
}

void cuda_print(cuda_t *cuda, FILE *file) {
    cuda_device_props(cuda);
    hipDeviceProp_t *prop = &cuda->props;

    fprintf(file, "Device Number: %d\n", cuda->devid);
    fprintf(file, "  Device name: %s\n", prop->name);
    fprintf(file, "  Memory Clock Rate (KHz): %d\n", prop->memoryClockRate);
    fprintf(file, "  Memory Bus Width (bits): %d\n", prop->memoryBusWidth);
    fprintf(file, "  Peak Memory Bandwidth (GB/s): %f\n",
            2.0 * prop->memoryClockRate * (prop->memoryBusWidth / 8) / 1.0e6);
    fprintf(file, "  Compute Version: %d.%d\n", prop->major, prop->minor);
    fprintf(file, "  Compute Mode: ");

    switch (prop->computeMode) {
        case hipComputeModeExclusive:
            fprintf(file, "Exclusive");
            break;
        case hipComputeModeProhibited:
            fprintf(file, "Prohibited");
            break;
        default:
            fprintf(file, "Default");
            break;
    }

    fprintf(file, "\n");
    fprintf(file, "  SM count: %d\n", prop->multiProcessorCount);
    fprintf(file, "  Shared mem/block: %zd\n", prop->sharedMemPerBlock);
    fprintf(file, "  Threads per warp: %d\n", prop->warpSize);
    fprintf(file, "  Max threads per block: %d\n", prop->maxThreadsPerBlock);

    fprintf(file, "  Max block size: (");
    for (int j = 0; j < 3; j++) {
        fprintf(file, "%d,", prop->maxThreadsDim[j]);
    }

    fprintf(file, ")\n  Max grid size: (");
    for (int j = 0; j < 3; j++) {
        fprintf(file, "%d,", prop->maxGridSize[j]);
    }

    fprintf(file, ")\n");
}

int cuda_set_device(cuda_t *cuda) {
    return hipSetDevice(cuda->devid);
}

int cuda_sync() {
    return hipDeviceSynchronize();
}

float cuda_profile_start(cuda_t *cuda) {
    gpuAssert(hipEventCreate(&cuda->start));
    gpuAssert(hipEventCreate(&cuda->stop));
    gpuAssert(hipEventRecord(cuda->start));
    return 0.0f;
}

float cuda_profile_stop(cuda_t *cuda) {
    float elapsed;
    hipDeviceSynchronize();
    gpuAssert(hipEventRecord(cuda->stop));
    gpuAssert(hipEventSynchronize(cuda->stop));
    gpuAssert(hipEventElapsedTime(&elapsed, cuda->start, cuda->stop));
    return elapsed * 1E-3f;
}

void* cuda_alloc(cuda_t *cuda, void** ptr, size_t size) {
    cuda->dataptr = cuda_malloc(ptr, size);
    cuda->datasize = size;
    return ptr;
}

void* cuda_alloc_man(cuda_t *cuda, void** ptr, size_t size) {
    if (ptr == NULL) {
        ptr = &cuda->dataptr;
    }
    gpuAssert(hipMallocManaged(&cuda->dataptr, size));
    gpuCheck("hipMallocManaged");
    cuda->datasize = size;
    cuda->managed = 1;
    return ptr;
}

void* cuda_malloc(void** ptr, size_t size) {
    gpuAssert(hipMalloc(ptr, size));
    gpuCheck("hipMalloc");
    return *ptr;
}

void cuda_free(void* ptr) {
    gpuAssert(hipFree(ptr));
}

void cuda_copy_host(const void* src, void* dest, size_t size) {
    fprintf(stderr, "src=%p, dest=%p,size=%u\n", src, dest, size);
    gpuAssert(hipMemcpy(dest, src, size, hipMemcpyDeviceToHost));
    gpuCheck("hipMemcpyDeviceToHost");
}

void cuda_copy_device(const void* src, void* dest, size_t size) {
    gpuAssert(hipMemcpy(dest, src, size, hipMemcpyHostToDevice));
    gpuCheck("hipMemcpyHostToDevice");
}
